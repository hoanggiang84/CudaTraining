#include "hip/hip_runtime.h"
#include <string.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "GpuTimer.h"

typedef float2 Complex;

#define NUM_STREAMS 4
#define NUM_RECORDS 2000
#define RECORD_LENGTH 500

__global__ void hit(hipfftComplex* d_matrix, int num_records, int record_length)
{
	int signalIndex = threadIdx.x; 
	if(signalIndex >= record_length)
		return;

	int recordIndex = blockIdx.x;
	int idx = recordIndex*record_length + signalIndex;
	float2 temp = d_matrix[idx];
	if(signalIndex <= record_length/2)
	{
		temp.x = temp.x * 2;
		temp.y = temp.y * 2;
		d_matrix[idx] = temp;
	}
	else if(signalIndex < record_length)
	{
		temp.x = 0;
		temp.y = 0;
		d_matrix[idx] = temp;
	}
}

__global__ void hft(hipfftComplex* d_matrix, hipfftComplex* d_original, int num_record, int record_length)
{
	int signalIndex = threadIdx.x; 
	if(signalIndex >= record_length)
		return;

	int recordIndex = blockIdx.x;
	int idx = recordIndex*record_length + signalIndex;
	//d_matrix[idx].x = d_original[idx].x ;
	d_matrix[idx].y = d_matrix[idx].y / record_length;
	d_matrix[idx].x = sqrtf(powf(d_original[idx].x,2) + powf(d_matrix[idx].y,2));
} 

__global__ void max_kernel(float * d_out, hipfftComplex * d_in)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Compare elements in first half with second half
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
			if(d_in[myId + s].x > d_in[myId].x)
				d_in[myId].x = d_in[myId + s].x;
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = d_in[myId].x;
    }
}

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/********/
/* MAIN */
/********/
int main()
{
	int sampleNum = NUM_RECORDS * RECORD_LENGTH;
	int mem_size = sizeof(Complex) * sampleNum;

	Complex* h_matrix = (Complex*)malloc(mem_size);
    for (int j=0; j<NUM_RECORDS; j++) 
	{
		for (int i=0; i<RECORD_LENGTH; i++) {
            h_matrix[j*RECORD_LENGTH+i].x = rand()/float(RAND_MAX);
			h_matrix[j*RECORD_LENGTH+i].y = 0;
        }
	}

	Complex* d_matrix;
	Complex* d_original;
	float* h_max = (float*)malloc(NUM_RECORDS*sizeof(float));
	float* d_max;

	gpuErrchk(hipHostRegister(h_matrix, mem_size, hipHostRegisterPortable));
	gpuErrchk(hipHostRegister(h_max, NUM_RECORDS*sizeof(float), hipHostRegisterPortable));
	gpuErrchk(hipMalloc((void**)&d_matrix, mem_size));
	gpuErrchk(hipMalloc((void**)&d_original, mem_size));
	gpuErrchk(hipMalloc((void**)&d_max, NUM_RECORDS*sizeof(float)));

	// --- Creates CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) 
		gpuErrchk(hipStreamCreate(&streams[i]));

    // --- Creates cuFFT plans and sets them in streams
    // --- Advanced data layout
    //     input[b * idist + x * istride]
    //     output[b * odist + x * ostride]
    //     b = signal number
    //     x = element of the b-th signal

    int rank = 1;                           // --- 1D FFTs
    int n[] = { RECORD_LENGTH };                        // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = RECORD_LENGTH, odist = RECORD_LENGTH;               // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = NUM_RECORDS/NUM_STREAMS;                      // --- Number of batched executions

    hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*NUM_STREAMS);
	int streamSize = sampleNum/NUM_STREAMS;
    for (int i = 0; i < NUM_STREAMS; i++) {
        //hipfftPlan1d(&plans[i], streamSize, HIPFFT_C2C, 1);
		hipfftPlanMany(&plans[i], rank, n, 
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2C, batch);
        hipfftSetStream(plans[i], streams[i]);
    }

	GpuTimer timer;
	timer.Start();

    // --- Async memcopyes and computations
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		int offset = i * streamSize;
		gpuErrchk(hipMemcpyAsync(&d_matrix[offset], &h_matrix[offset], streamSize*sizeof(float2), hipMemcpyHostToDevice, streams[i]));
		gpuErrchk(hipMemcpyAsync(&d_original[offset], &h_matrix[offset], streamSize*sizeof(float2), hipMemcpyHostToDevice, streams[i]));
	}
    
	int blockSize = RECORD_LENGTH;
	int gridSize = batch;
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		int offset = i * streamSize;
		hipfftExecC2C(plans[i], (hipfftComplex*)(&d_matrix[offset]), (hipfftComplex*)(&d_matrix[offset]), HIPFFT_FORWARD);
		hit<<<gridSize, blockSize, 0, streams[i]>>>((hipfftComplex*)(&d_matrix[offset]), streamSize, RECORD_LENGTH);
		hipfftExecC2C(plans[i], (hipfftComplex*)(&d_matrix[offset]), (hipfftComplex*)(&d_matrix[offset]), HIPFFT_BACKWARD);
		hft<<<gridSize, blockSize, 0, streams[i]>>>((hipfftComplex*)(&d_matrix[offset]), (hipfftComplex*)(&d_original[offset]), streamSize, RECORD_LENGTH);

		int offset2 = i * batch;
		max_kernel<<<gridSize, blockSize, 0, streams[i]>>>((float*)(&d_max[offset2]), (hipfftComplex*)(&d_matrix[offset]));
	}

	for (int i = 0; i < NUM_STREAMS; i++)
	{
		int offset = i * streamSize;
		gpuErrchk(hipMemcpyAsync(&h_matrix[offset], &d_matrix[offset], streamSize*sizeof(float2), hipMemcpyDeviceToHost, streams[i]));

		int offset2 = i * batch;
		gpuErrchk(hipMemcpyAsync(&h_max[offset2], &d_max[offset2], batch*sizeof(float), hipMemcpyDeviceToHost, streams[i]));
	}

    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamSynchronize(streams[i]));

	timer.Stop();
	float ms = timer.Elapsed();
   
	hipFree(d_matrix);
	hipFree(d_original);
	hipFree(d_max);
	hipHostUnregister(h_matrix);
	hipHostUnregister(h_max);
	free(h_matrix);
	free(h_max);

	for(int i = 0; i < NUM_STREAMS; i++) 
		gpuErrchk(hipStreamDestroy(streams[i]));

	printf("Stream Fast Fourier Transform. Time Elapsed: %fms", ms);
	printf("\nPress any key to exit...");
    char c;
    scanf("%c", &c);

    return 0;
}